#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
#include "Timer.h"
using namespace std;

struct Image {
	unsigned char* data;
	int* dataCompressed;
	int width;
	int height;
	int nrChannels;
};
//
//Image sample;
//Image input;
//Image output;

Timer timer;

unsigned char* host_sample;
unsigned char* host_input;
unsigned char* host_output;
int host_sampleWidth;
int host_sampleHeight;
int host_sampleChannels;
int host_inputWidth;
int host_inputHeight;
int host_inputChannels;
int host_NSize;
float* host_distances;

int* host_sampleC;
int* host_inputC;
int* host_outputC;

unsigned char* dev_sample;
unsigned char* dev_input;
unsigned char* dev_output;
int* dev_sampleWidth;
int* dev_sampleHeight;
int* dev_sampleChannels;
int* dev_inputWidth;
int* dev_inputHeight;
int* dev_inputChannels;
int* dev_NSize;
int* dev_rx;
int* dev_ry;
float* dev_distances;

int* dev_sampleC;
int* dev_inputC;
int* dev_outputC;

int *dev_outputH;


//void LoadImage(const char* path, Image* img) {
//
//	img->data = stbi_load(path, &img->width, &img->height, &img->nrChannels, 0);
//}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	//if (code != hipSuccess)
	//{
	fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	//if (abort) exit(code);
	//}
}

float DIST(int c0, int c1) {
	int r1 = (c0 >> 24) & 0xff;
	int g1 = (c0 >> 16) & 0xff;
	int b1 = (c0 >> 8) & 0xff;
	int a1 = c0 & 0xff;

	int r2 = (c1 >> 24) & 0xff;
	int g2 = (c1 >> 16) & 0xff;
	int b2 = (c1 >> 8) & 0xff;
	int a2 = c1 & 0xff;

	/*cout << "Desempacotando bits: " << endl;
	cout << "red: " << r1 << endl;
	cout << "blue: " << g1 << endl;
	cout << "green: " << b1 << endl;
	cout << "alpha: " << a1 << endl;*/

	int r = r1 - r2;
	int g = g1 - g2;
	int b = b1 - b2;

	return sqrt(r * r + g * g + b * b);
}

void EmpacotarBits(Image* img) {
	int j = 0;

	//cout << "Empacotando bits de: " << img << endl;

	for (int i = 0; i < img->height * img->width * img->nrChannels; i += img->nrChannels)
	{
		int r = img->data[i];
		int g = img->data[i + 1];
		int b = img->data[i + 2];
		int a = img->data[i + 3];
		int rgba = (r << 24) | (g << 16) | (b << 8) | (a);
		img->dataCompressed[j] = rgba;
		//cout << rgba << endl;
		j++;
	}
}

void EmpacotarBits(unsigned char* data, int* dest, int w, int h, int ch) {
	int j = 0;

	//cout << "Empacotando bits de: " << img << endl;

	for (int i = 0; i < h * w * ch; i += ch)
	{
		int r = data[i];
		int g = data[i + 1];
		int b = data[i + 2];
		int a = data[i + 3];
		int rgba = (r << 24) | (g << 16) | (b << 8) | (a);
		dest[j] = rgba;
		//cout << rgba << endl;
		j++;
	}
}

void DesempacotarBits(int* data, unsigned char* dest, int w, int h, int ch) {

	int j = 0;

	for (int i = 0; i < w * h; i++)
	{
		int rgba = data[i];
		int r1 = (rgba >> 24) & 0xff;
		int g1 = (rgba >> 16) & 0xff;
		int b1 = (rgba >> 8) & 0xff;
		int a1 = rgba & 0xff;

		dest[j] = r1;
		dest[j + 1] = g1;
		dest[j + 2] = b1;
		dest[j + 3] = a1;

		j += ch;
	}
}

void DesempacotarBits(Image* img) {

	int j = 0;

	for (int i = 0; i < img->height * img->width; i++)
	{
		int rgba = img->dataCompressed[i];
		int r1 = (rgba >> 24) & 0xff;
		int g1 = (rgba >> 16) & 0xff;
		int b1 = (rgba >> 8) & 0xff;
		int a1 = rgba & 0xff;

		img->data[j] = r1;
		img->data[j + 1] = g1;
		img->data[j + 2] = b1;
		img->data[j + 3] = a1;

		j += img->nrChannels;
	}
}

void Init() {
	/*LoadImage("input.png", &input);
	LoadImage("sample64.png", &sample);*/

	host_input = stbi_load("input.png", &host_inputWidth, &host_inputHeight, &host_inputChannels, 0);
	host_sample = stbi_load("sample64.png", &host_sampleWidth, &host_sampleHeight, &host_sampleChannels, 0);

	/*input.dataCompressed = new int[input.width * input.height];
	output.dataCompressed = new int[output.width * output.height];
	sample.dataCompressed = new int[sample.width * sample.height];*/

	host_inputC = new int[host_inputWidth * host_inputHeight];
	host_outputC = new int[host_inputWidth * host_inputHeight];
	host_sampleC = new int[host_sampleWidth * host_sampleHeight];

	/*EmpacotarBits(&input);
	EmpacotarBits(&sample);*/

	EmpacotarBits(host_input, host_inputC, host_inputWidth, host_inputHeight, host_inputChannels);
	EmpacotarBits(host_sample, host_sampleC, host_sampleWidth, host_sampleHeight, host_sampleChannels);

	/*output.height = input.height;
	output.width = input.width;
	output.nrChannels = input.nrChannels;
	output.data = input.data;
	output.dataCompressed = input.dataCompressed;*/

	host_output = host_input;
	host_outputC = host_inputC;

	//alocar mem�ria
	host_NSize = 6;
	int n1 = host_sampleHeight * host_sampleWidth;
	int n2 = host_inputWidth * host_inputHeight;

	host_distances = new float[n1];

	gpuErrchk(hipMallocManaged(&dev_sampleC, n1 * sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_outputC, n2 * sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_sampleHeight, sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_sampleWidth, sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_NSize, sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_inputWidth, sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_outputH, sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_rx, sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_ry, sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_distances, n1 * sizeof(float)));

	//copy memory
	gpuErrchk(hipMemcpy(dev_sampleC, host_sampleC, n1 * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_outputC, host_inputC, n2 * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_sampleHeight, &host_sampleHeight, sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_sampleWidth, &host_sampleWidth, sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_NSize, &host_NSize, sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_inputWidth, &host_inputWidth, sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_outputH, &host_inputHeight, sizeof(int), hipMemcpyHostToDevice));
}

int GetVizinhoSample(int positionX, int positionY, int neighborPosX, int neighborPoxY) {
	int pos = 0;
	int posx = 0;
	int posy = 0;

	posx = positionX + neighborPosX;
	posy = positionY + neighborPoxY;

	if (posx < 0 || posy < 0)
		return 0;

	pos = posx + posy * host_sampleWidth;
	return host_sampleC[pos];
}

int GetVizinhoInput(int positionX, int positionY, int neighborPosX, int neighborPoxY) {
	int pos = 0;
	int posx = 0;
	int posy = 0;

	posx = positionX + neighborPosX;
	posy = positionY + neighborPoxY;

	if (posx < 0 || posy < 0)
		return 0;

	pos = posx + posy * host_inputWidth;
	return host_outputC[pos];
}
void SaveImage() {
	//stbi_write_jpg("test1.jpg", sample.width, sample.height, sample.nrChannels, sample.data, sample.width * sample.height * sample.nrChannels);
	stbi_write_jpg("output.jpg", host_inputWidth, host_inputHeight, host_inputChannels, host_output, host_inputWidth* host_inputHeight* host_inputChannels);
}

__global__
void DoTexture(int* sampleDataC, int* outputDataC, int rx, int ry, int sampleH, int sampleW, int NSize, int outputW, int outputH, float* dist)
{
	unsigned int dmin = 999999;
	int pixel = 0;
	int pos = 0;
	int posx = 0;
	int posy = 0;
	float d = 0;

	for (int y = -NSize; y <= NSize; y++) {
		for (int x = -NSize; x <= NSize; x++) {
			int ss;
			posx = threadIdx.x + x;
			posy = blockIdx.x + y;

			/*int a = threadIdx.x + blockIdx.x * sampleW;
			pos = (a + x + sampleW) % sampleW;
			ss = sampleDataC[pos];*/

			if (posx < 0 || posy < 0) {
			ss = 0;
			}
			else {
			pos = posx + posy * sampleW;
			ss = sampleDataC[pos];
			}

			int rr;
			pos = 0;
			/*a = rx + ry*outputW;
			pos = (a + y + sampleH) % outputH;
			rr = outputDataC[pos];*/
			posx = 0;
			posy = 0;

			posx = rx + x;
			posy = ry + y;

			if (posx < 0 || posy < 0) {
			rr = 0;
			}
			else {
			pos = posx + posy * outputW;
			rr = outputDataC[pos];
			}

			int r1 = (ss >> 24) & 0xff;
			int g1 = (ss >> 16) & 0xff;
			int b1 = (ss >> 8) & 0xff;
			int a1 = ss & 0xff;
			int r2 = (rr >> 24) & 0xff;
			int g2 = (rr >> 16) & 0xff;
			int b2 = (rr >> 8) & 0xff;
			int a2 = rr & 0xff;
			int r = r1 - r2;
			int g = g1 - g2;
			int b = b1 - b2;

			d += sqrtf(r * r + g * g + b * b);
		}
	}
	int arrayPos = threadIdx.x + blockIdx.x * sampleW;
	dist[arrayPos] = d;
}

void CreateTexture() {
	int rx = 0;
	int ry = 0;
	float d = 0;
	int pixel = 0;
	for (ry = 0; ry < host_inputHeight; ry++) {
		for (rx = 0; rx < host_inputWidth; rx++) {
			float dmin = 999999;
			gpuErrchk(hipMemcpy(dev_rx, &rx, sizeof(int), hipMemcpyHostToDevice));
			gpuErrchk(hipMemcpy(dev_ry, &ry, sizeof(int), hipMemcpyHostToDevice));

			DoTexture << <host_sampleWidth, host_sampleHeight >> > (dev_sampleC, dev_outputC, *dev_rx, *dev_ry, *dev_sampleHeight, *dev_sampleWidth, *dev_NSize, *dev_inputWidth, *dev_outputH, dev_distances);
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());

			int n1 = host_sampleHeight * host_sampleWidth;
			gpuErrchk(hipMemcpy(host_distances, dev_distances, n1 * sizeof(float), hipMemcpyDeviceToHost));


			for (int i = 0; i <n1; i++)
			{
				d = host_distances[i];
				if (d < dmin) {
					pixel = host_sampleC[i];
					dmin = d;
				}
			}

			int pos = rx + ry * host_inputWidth;
			host_outputC[pos] = pixel;
		}
	}


	//gpuErrchk(hipMemcpy(host_outputC, dev_outputC, n2 * sizeof(int), hipMemcpyDeviceToHost));

	/*int n2 = 6 / 2;
	int pos = 0;
	int pixel = 0;
	for (int ry = 0; ry < host_inputHeight; ry++) {
	for (int rx = 0; rx < host_inputWidth; rx++) {

	unsigned int dmin = 999999;

	for (int sy = 0; sy < sample.height; sy++) {
	for (int sx = 0; sx < sample.width; sx++) {

	float d = 0;

	for (int y = -n2; y <= n2; y++) {
	for (int x = -n2; x <= n2; x++) {
	int s = GetVizinhoSample(sx, sy, x, y);
	int r = GetVizinhoInput(rx, ry, x, y);
	d += DIST(r, s);
	}
	}
	if (d < dmin) {
	pixel = sample.dataCompressed[sx + sy * sample.width];
	dmin = d;
	}
	}

	}
	pos = rx + ry * output.width;
	output.dataCompressed[pos] = pixel;
	}

	}*/
}

int main()
{
	timer.start();
	Init();
	CreateTexture();
	DesempacotarBits(host_outputC, host_output, host_inputWidth, host_inputHeight, host_inputChannels);
	SaveImage();
	timer.finish();
	cout << "Tempo do algoritmo em milisegundos: "<<timer.getElapsedTimeMs()<<endl;

	system("PAUSE");
	return 0;
}